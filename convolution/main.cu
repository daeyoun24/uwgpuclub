#include "hip/hip_runtime.h"
/**
 * Daeyoun Kim
 * GitHub: https://github.com/daeyoun24/uwgpuclub
 */


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define WIDTH 244
#define TILE_WIDTH 32
#define MASK_WIDTH 31
#define H_MASK_WIDTH 31 / 2

__constant__ float M[MASK_WIDTH][MASK_WIDTH];

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 *
 * See hip/hip_runtime.h for error code descriptions.
 */
#define CHECK_CUDA_RESULT(N) {											\
	if (N != 0) {														\
		printf("CUDA call on line %d returned error %d\n", __LINE__,N);	\
		exit(1);														\
	} }

__global__ void convKernel(float *A, float *B) {
	unsigned int ty = threadIdx.y;
	unsigned int tx = threadIdx.x;
	unsigned int row = blockIdx.y * blockDim.y + ty;
	unsigned int col = blockIdx.x * blockDim.x + tx;

	__shared__ float s_A[TILE_WIDTH + MASK_WIDTH / 2][TILE_WIDTH + MASK_WIDTH / 2];

	if (row < WIDTH && col < WIDTH) {
		// Top left
		if (row - H_MASK_WIDTH < 0 && col - H_MASK_WIDTH < 0) {
			s_A[ty][tx] = 0;
		}
		// Top
		else if (row - H_MASK_WIDTH < 0) {
			s_A[ty][tx + H_MASK_WIDTH] = 0;
		}
		// Top right
		else if (row - H_MASK_WIDTH < 0 && col + H_MASK_WIDTH >= WIDTH) {
			s_A[ty][tx + H_MASK_WIDTH * 2] = 0;
		}
		// Right
		else if (col + H_MASK_WIDTH >= WIDTH) {
			s_A[ty + H_MASK_WIDTH][tx + H_MASK_WIDTH * 2] = 0;
		}
		// Bottom right
		else if (row + H_MASK_WIDTH >= WIDTH && col + H_MASK_WIDTH >= WIDTH) {
			s_A[ty + H_MASK_WIDTH * 2][tx + H_MASK_WIDTH * 2] = 0;
		}
		// Bottom
		else if (row + H_MASK_WIDTH >= WIDTH) {
			s_A[ty + H_MASK_WIDTH * 2][tx + H_MASK_WIDTH] = 0;
		}
		// Bottom left
		else if (row + H_MASK_WIDTH >= WIDTH && col - H_MASK_WIDTH < 0) {
			s_A[ty + H_MASK_WIDTH * 2][tx] = 0;
		}
		// Left
		else if (col - H_MASK_WIDTH < 0) {
			s_A[ty + H_MASK_WIDTH][tx] = 0;
		}
		// Center
		else {
			s_A[ty + H_MASK_WIDTH][tx + H_MASK_WIDTH] = A[row * WIDTH + col];
		}

		__syncthreads();
	}
}

int main()
{
	float *M_h, *A_h, *B_h, *A_d, *B_d;

	M_h = (float *)malloc(MASK_WIDTH * MASK_WIDTH * sizeof(float));
	A_h = (float *)malloc(WIDTH * WIDTH * sizeof(float));
	B_h = (float *)malloc(WIDTH * WIDTH * sizeof(float));

	srand(time(NULL));

	for (unsigned int i = 0; i < MASK_WIDTH * MASK_WIDTH; ++i) {
		M_h[i] = (float)rand() / RAND_MAX * 5.0f;
	}

	CHECK_CUDA_RESULT(hipMemcpyToSymbol(HIP_SYMBOL(M), M_h, (MASK_WIDTH * MASK_WIDTH) * sizeof(float)));

	for (unsigned int i = 0; i < WIDTH * WIDTH; ++i) {
		A_h[i] = (float)rand() % 256;
	}

	CHECK_CUDA_RESULT(hipMalloc(&A_d, WIDTH * WIDTH * sizeof(float)));
	CHECK_CUDA_RESULT(hipMalloc(&B_d, WIDTH * WIDTH * sizeof(float)));

	CHECK_CUDA_RESULT(hipMemcpy(A_d, A_h, WIDTH * WIDTH * sizeof(float), hipMemcpyHostToDevice));

	dim3 dimBlock (TILE_WIDTH, TILE_WIDTH, 1);
	dim3 dimGrid (ceil((double)WIDTH / TILE_WIDTH), ceil((double)WIDTH / TILE_WIDTH), 1);

	convKernel<<<dimBlock, dimGrid>>>(A_d, B_d);

	CHECK_CUDA_RESULT(hipMemcpy(B_h, B_d, WIDTH * WIDTH * sizeof(float), hipMemcpyDeviceToHost));

	// Verify the results here ...

	hipFree(A_d);
	hipFree(B_d);
	free(A_h);
	free(B_h);
	free(M_h);

	return 0;
}
